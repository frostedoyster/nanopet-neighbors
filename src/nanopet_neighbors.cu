#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <cmath>


std::vector<torch::Tensor> get_nef_indices(
    torch::Tensor centers,
    int64_t n_nodes,
    int64_t n_edges_per_node
) {
    torch::Device original_device = centers.device();
    centers = centers.to(torch::kCPU);

    centers = centers.to(torch::kLong).contiguous();

    int64_t n_edges = centers.size(0);
    torch::Tensor edges_to_nef = torch::zeros(
        {n_nodes, n_edges_per_node}, torch::TensorOptions().dtype(torch::kLong).device(centers.device())
    );
    torch::Tensor nef_to_edges_neighbor = torch::empty(
        {n_edges}, torch::TensorOptions().dtype(torch::kLong).device(centers.device())
    );
    std::vector<long> node_counter(n_nodes, 0);
    torch::Tensor nef_mask = torch::full(
        {n_nodes, n_edges_per_node}, 0, torch::TensorOptions().dtype(torch::kBool).device(centers.device())
    );

    long* centers_ptr = centers.data_ptr<long>();
    long* edges_to_nef_ptr = edges_to_nef.data_ptr<long>();
    long* nef_to_edges_neighbor_ptr = nef_to_edges_neighbor.data_ptr<long>();
    bool* nef_mask_ptr = nef_mask.data_ptr<bool>();

    for (int64_t i = 0; i < n_edges; i++) {
        long center = centers_ptr[i];
        edges_to_nef_ptr[center * n_edges_per_node + node_counter[center]] = i;
        nef_mask_ptr[center * n_edges_per_node + node_counter[center]] = true;
        nef_to_edges_neighbor_ptr[i] = node_counter[center];
        node_counter[center] += 1;
    }

    edges_to_nef = edges_to_nef.to(original_device);
    nef_to_edges_neighbor = nef_to_edges_neighbor.to(original_device);
    nef_mask = nef_mask.to(original_device);

    return {edges_to_nef, nef_to_edges_neighbor, nef_mask};
}


__global__ void find_corresponding_edges_kernel(
    const long* centers_ptr,
    const long* neighbors_ptr,
    const long* shift_x_ptr,
    const long* shift_y_ptr,
    const long* shift_z_ptr,
    long* inverse_indices_ptr,
    int64_t n_edges
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n_edges) {
        bool found = false;
        for (int64_t j = 0; j < n_edges; j++) {
            if (centers_ptr[i] == neighbors_ptr[j] && centers_ptr[j] == neighbors_ptr[i] && shift_x_ptr[i] == -shift_x_ptr[j] && shift_y_ptr[i] == -shift_y_ptr[j] && shift_z_ptr[i] == -shift_z_ptr[j]) {
                inverse_indices_ptr[i] = j;
                found = true;
                break;
            }
        }
        if (!found) {
            inverse_indices_ptr[i] = -1; // Use -1 to indicate no corresponding edge found
        }
    }
}

torch::Tensor get_corresponding_edges(
    torch::Tensor array
) {
    torch::Tensor centers = array.index({torch::indexing::Slice(), 0}).to(torch::kLong).contiguous();
    torch::Tensor neighbors = array.index({torch::indexing::Slice(), 1}).to(torch::kLong).contiguous();
    torch::Tensor shift_x = array.index({torch::indexing::Slice(), 2}).to(torch::kLong).contiguous();
    torch::Tensor shift_y = array.index({torch::indexing::Slice(), 3}).to(torch::kLong).contiguous();
    torch::Tensor shift_z = array.index({torch::indexing::Slice(), 4}).to(torch::kLong).contiguous();

    long* centers_ptr = centers.data_ptr<long>();
    long* neighbors_ptr = neighbors.data_ptr<long>();
    long* shift_x_ptr = shift_x.data_ptr<long>();
    long* shift_y_ptr = shift_y.data_ptr<long>();
    long* shift_z_ptr = shift_z.data_ptr<long>();

    int64_t n_edges = centers.size(0);

    torch::Tensor inverse_indices = torch::empty(
        {n_edges}, torch::TensorOptions().dtype(torch::kLong).device(centers.device())
    );
    long* inverse_indices_ptr = inverse_indices.data_ptr<long>();

    int threads_per_block = 256;
    int num_blocks = (n_edges + threads_per_block - 1) / threads_per_block;

    find_corresponding_edges_kernel<<<num_blocks, threads_per_block>>>(
        centers_ptr,
        neighbors_ptr,
        shift_x_ptr,
        shift_y_ptr,
        shift_z_ptr,
        inverse_indices_ptr,
        n_edges
    );

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throw std::runtime_error(std::string("CUDA kernel failed: ") + hipGetErrorString(err));
    }

    if (torch::any(inverse_indices == -1).item<bool>()) {
        throw std::runtime_error("Some edges do not have corresponding edges");
    }

    return inverse_indices;
}


TORCH_LIBRARY(nanopet_neighbors_cuda, m) {
    m.def(
        "get_nef_indices",
         &get_nef_indices
    );
    m.def(
        "get_corresponding_edges",
         &get_corresponding_edges
    );
}
